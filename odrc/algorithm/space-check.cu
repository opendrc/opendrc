#include "hip/hip_runtime.h"
#include <odrc/algorithm/parallel_mode.hpp>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <unordered_map>
#include <unordered_set>
#include <vector>

#include <thrust/async/sort.h>
#include <thrust/device_reference.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include <odrc/algorithm/layout-partition.hpp>
#include <odrc/core/interval_tree.hpp>

namespace odrc {

using coord    = odrc::core::coord;
using polygon  = odrc::core::polygon;
using cell_ref = odrc::core::cell_ref;
using o_edge   = odrc::core::orthogonal_edge;

__global__ void hcheck_kernel(o_edge*       h_edges,
                              int           his,
                              int           hie,
                              int           hjs,
                              int           hje,
                              int           threshold,
                              check_result* results,
                              int*          res_offset) {
  int tid     = threadIdx.x + blockIdx.x + blockDim.x;
  int total_i = hie - his;
  int total_j = hje - hjs;
  if (tid >= total_i * total_j)
    return;
  int  i            = tid / total_j;
  int  j            = tid % total_j;
  int  e11x         = h_edges[i].p_start;
  int  e11y         = h_edges[i].intercept;
  int  e12x         = h_edges[i].p_end;
  int  e12y         = h_edges[i].intercept;
  int  e21x         = h_edges[j].p_start;
  int  e21y         = h_edges[j].intercept;
  int  e22x         = h_edges[j].p_end;
  int  e22y         = h_edges[j].intercept;
  bool is_violation = false;

  if (e11y < e22y) {
    // e22 e21
    // e11 e12
    bool is_outside_to_outside = e11x < e12x and e21x > e22x;
    bool is_too_close          = e21y - e11y < threshold;
    bool is_projection_overlap = e21x < e11x and e12x < e22x;
    is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
  } else {
    // e12 e11
    // e21 e22
    bool is_outside_to_outside = e21x < e22x and e11x > e12x;
    bool is_too_close          = e11y - e21y < threshold;
    bool is_projection_overlap = e11x < e21x and e22x < e12x;
    is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
  }
  return;
  if (is_violation) {
    check_result& res = results[atomicAdd(res_offset, 1)];
    res.e11x          = e11x;
    res.e11y          = e11y;
    res.e12x          = e12x;
    res.e12y          = e12y;
    res.e21x          = e21x;
    res.e21y          = e21y;
    res.e22x          = e22x;
    res.e22y          = e22y;
    res.is_violation  = false;
  }
}

__global__ void vcheck_kernel(o_edge*       v_edges,
                              int           vis,
                              int           vie,
                              int           vjs,
                              int           vje,
                              int           threshold,
                              check_result* results,
                              int*          res_offset) {
  int tid     = threadIdx.x + blockIdx.x + blockDim.x;
  int total_i = vie - vis;
  int total_j = vje - vjs;
  if (tid >= total_i * total_j)
    return;
  int  i            = tid / total_j;
  int  j            = tid % total_j;
  int  e11x         = v_edges[i].intercept;
  int  e11y         = v_edges[i].p_start;
  int  e12x         = v_edges[i].intercept;
  int  e12y         = v_edges[i].p_end;
  int  e21x         = v_edges[j].intercept;
  int  e21y         = v_edges[j].p_start;
  int  e22x         = v_edges[j].intercept;
  int  e22y         = v_edges[j].p_end;
  bool is_violation = false;

  if (e11x < e21x) {
    // e11 e22
    // e12 e21
    bool is_outside_to_outside = e11y > e12y and e21y < e22y;
    bool is_too_close          = e21x - e11x < threshold;
    bool is_projection_overlap = e11y < e21y and e22y < e12y;
    is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;

  } else {
    // e21 e12
    // e22 e11
    bool is_outside_to_outside = e21y > e22y and e11y < e21y;
    bool is_too_close          = e11x - e21x < threshold;
    bool is_projection_overlap = e21y < e11y and e12y < e22y;
    is_violation =
        is_outside_to_outside and is_too_close and is_projection_overlap;
  }

  if (is_violation) {
    check_result& res = results[atomicAdd(res_offset, 1)];
    res.e11x          = e11x;
    res.e11y          = e11y;
    res.e12x          = e12x;
    res.e12y          = e12y;
    res.e21x          = e21x;
    res.e21y          = e21y;
    res.e22x          = e22x;
    res.e22y          = e22y;
    res.is_violation  = false;
  }
}

__device__ inline void run_check(o_edge*       h_edges,
                                 o_edge*       v_edges,
                                 int*          h_idx,
                                 int*          v_idx,
                                 int*          mbrs_d,
                                 int           i,
                                 int           j,
                                 int           tid,
                                 int           threshold,
                                 int*          vio_offset,
                                 check_result* check_results,
                                 o_edge*       hbuf,
                                 o_edge*       vbuf) {
  int sum = 0;

  // if (i >= csize or j >= csize) {
  //   printf("thread %d check %d %d", tid, i, j);
  // }
  int jxmin = mbrs_d[j * 4];
  int jxmax = mbrs_d[j * 4 + 1];
  int jymin = mbrs_d[j * 4 + 2];
  int jymax = mbrs_d[j * 4 + 3];
  if (mbrs_d[j * 4] > mbrs_d[i * 4 + 1] or mbrs_d[j * 4 + 1] < mbrs_d[i * 4] or
      mbrs_d[j * 4 + 2] > mbrs_d[i * 4 + 3] or
      mbrs_d[j * 4 + 3] < mbrs_d[i * 4 + 2])
    return;

  int his = h_idx[i];
  int hie = h_idx[i + 1];
  int hjs = h_idx[j];
  int hje = h_idx[j + 1];
  int vis = v_idx[i];
  int vie = v_idx[i + 1];
  int vjs = v_idx[j];
  int vje = v_idx[j + 1];
  for (int i = 0; i < 50; ++i) {
    if (i >= hje)
      break;
    hbuf[i * blockDim.x + threadIdx.x] = h_edges[hjs + i];
    vbuf[i * blockDim.x + threadIdx.x] = v_edges[vjs + i];
  }

  for (int i = hie; i < his; ++i) {
    int iy = h_edges[i].intercept;
    if (iy - threshold >= jymax)
      break;
    else if (iy + threshold <= jymin)
      continue;
    int e11x = h_edges[i].p_start;
    int e11y = h_edges[i].intercept;
    int e12x = h_edges[i].p_end;
    int e12y = h_edges[i].intercept;
    for (int j = 0; j < hje - hjs; ++j) {
      int jy = h_edges[j].intercept;
      if (jy - threshold >= iy)
        break;
      else if (jy + threshold <= iy)
        continue;
      int  e21x         = hbuf[j * blockDim.x + threadIdx.x].p_start;
      int  e21y         = hbuf[j * blockDim.x + threadIdx.x].intercept;
      int  e22x         = hbuf[j * blockDim.x + threadIdx.x].p_end;
      int  e22y         = hbuf[j * blockDim.x + threadIdx.x].intercept;
      bool is_violation = false;

      if (e11y < e22y) {
        // e22 e21
        // e11 e12
        bool is_outside_to_outside = e11x < e12x and e21x > e22x;
        bool is_too_close          = e21y - e11y < threshold;
        bool is_projection_overlap = e21x < e11x and e12x < e22x;
        is_violation =
            is_outside_to_outside and is_too_close and is_projection_overlap;
      } else {
        // e12 e11
        // e21 e22
        bool is_outside_to_outside = e21x < e22x and e11x > e12x;
        bool is_too_close          = e11y - e21y < threshold;
        bool is_projection_overlap = e11x < e21x and e22x < e12x;
        is_violation =
            is_outside_to_outside and is_too_close and is_projection_overlap;
      }
      if (is_violation) {
        int           offset = atomicAdd(vio_offset, 1);
        check_result& res    = check_results[offset];
        res.e11x             = e11x;
        res.e11y             = e11y;
        res.e12x             = e12x;
        res.e12y             = e12y;
        res.e21x             = e21x;
        res.e21y             = e21y;
        res.e22x             = e22x;
        res.e22y             = e22y;
        res.is_violation     = false;
      }
    }
  }
  for (int i = vie; i < vis; ++i) {
    int ix = v_edges[i].intercept;
    if (ix - threshold >= jxmax)
      break;
    else if (ix + threshold <= jxmin)
      continue;
    int e11x = v_edges[i].intercept;
    int e11y = v_edges[i].p_start;
    int e12x = v_edges[i].intercept;
    int e12y = v_edges[i].p_end;
    for (int j = 0; j < vje - vjs; ++j) {
      int jx = v_edges[j].intercept;
      if (jx - threshold >= ix)
        break;
      else if (jx + threshold <= ix)
        continue;
      int  e21x         = vbuf[j * blockDim.x + threadIdx.x].intercept;
      int  e21y         = vbuf[j * blockDim.x + threadIdx.x].p_start;
      int  e22x         = vbuf[j * blockDim.x + threadIdx.x].intercept;
      int  e22y         = vbuf[j * blockDim.x + threadIdx.x].p_end;
      bool is_violation = false;
      if (e11x < e21x) {
        // e11 e22
        // e12 e21
        bool is_outside_to_outside = e11y > e12y and e21y < e22y;
        bool is_too_close          = e21x - e11x < threshold;
        bool is_projection_overlap = e11y < e21y and e22y < e12y;
        is_violation =
            is_outside_to_outside and is_too_close and is_projection_overlap;
      } else {
        // e21 e12
        // e22 e11
        bool is_outside_to_outside = e21y > e22y and e11y < e21y;
        bool is_too_close          = e11x - e21x < threshold;
        bool is_projection_overlap = e21y < e11y and e12y < e22y;
        is_violation =
            is_outside_to_outside and is_too_close and is_projection_overlap;
      }

      if (is_violation) {
        int           offset = atomicAdd(vio_offset, 1);
        check_result& res    = check_results[offset];
        res.e11x             = e11x;
        res.e11y             = e11y;
        res.e12x             = e12x;
        res.e12y             = e12y;
        res.e21x             = e21x;
        res.e21y             = e21y;
        res.e22x             = e22x;
        res.e22y             = e22y;
        res.is_violation     = false;
      }
    }
  }
}

__global__ void run_row(o_edge*       h_edges,
                        o_edge*       v_edges,
                        int*          h_idx,
                        int*          v_idx,
                        int*          mbrs_d,
                        check_result* results,
                        evnt*         events,
                        int*          eidx,
                        int           nrows,
                        int           threshold,
                        int           start) {
  __shared__ int    vio_offset;
  __shared__ o_edge hbuf[32 * 50];
  __shared__ o_edge vbuf[32 * 50];
  int               tid = threadIdx.x + blockDim.x * blockIdx.x * 2 + start;
  if (tid >= nrows)
    return;
  if (tid == 0) {
    vio_offset = 0;
  }
  struct idv {
    int  id;
    bool is_valid = false;
  } idvs[10];
  int rs = eidx[tid];
  int re = eidx[tid + 1];
  __syncthreads();
  for (int i = rs; i < re; ++i) {
    evnt& e = events[i];
    if (e.id > 0) {  // check and insert
      int  eid  = e.id - 1;
      int* mbre = &mbrs_d[eid * 4];
      // self check
      run_check(h_edges, v_edges, h_idx, v_idx, mbrs_d, eid, eid, tid,
                threshold, &vio_offset, results, hbuf, vbuf);

      bool found = false;
      for (int j = 0; j < 10; ++j) {
        if (!idvs[j].is_valid) {  // place or skip
          if (found) {
            continue;
          }
          idvs[j].id       = eid;
          idvs[j].is_valid = true;
          found            = true;
        } else {
          run_check(h_edges, v_edges, h_idx, v_idx, mbrs_d, eid, idvs[j].id,
                    tid, threshold, &vio_offset, results, hbuf, vbuf);
        }
      }
    } else {
      for (int j = 0; j < 10; ++j) {
        if (idvs[j].id == -e.id - 1 and idvs[j].is_valid) {
          idvs[j].is_valid = false;
          break;
        }
      }
    }
  }
}

void space_check_par(odrc::core::database&         db,
                     int                           layer1,
                     int                           threshold,
                     std::vector<core::violation>& vios) {
  const auto&         cell_refs = db.get_top_cell().cell_refs;
  std::vector<int>    cells;
  std::vector<o_edge> hes;
  std::vector<o_edge> ves;
  std::vector<int>    hidx;
  std::vector<int>    vidx;
  std::vector<int>    mbrs;
  cells.reserve(cell_refs.size());
  for (int i = 0; i < cell_refs.size(); ++i) {
    const auto& cr       = cell_refs[i];
    const auto& the_cell = db.get_cell(cr.cell_name);
    if (!the_cell.is_touching(layer1)) {
      continue;
    }
    cells.emplace_back(i);
    hidx.emplace_back(hes.size());
    vidx.emplace_back(ves.size());
    hes.insert(hes.end(), cr.left_edges.at(layer1).begin(),
               cr.left_edges.at(layer1).end());
    hes.insert(hes.end(), cr.right_edges.at(layer1).begin(),
               cr.right_edges.at(layer1).end());
    ves.insert(ves.end(), cr.lower_edges.at(layer1).begin(),
               cr.lower_edges.at(layer1).end());
    ves.insert(ves.end(), cr.upper_edges.at(layer1).begin(),
               cr.upper_edges.at(layer1).end());
    mbrs.emplace_back(cr.cell_ref_mbr.x_min);
    mbrs.emplace_back(cr.cell_ref_mbr.x_max);
    mbrs.emplace_back(cr.cell_ref_mbr.y_min);
    mbrs.emplace_back(cr.cell_ref_mbr.y_max);
  }
  hidx.emplace_back(hes.size());
  vidx.emplace_back(ves.size());
  hipStream_t stream1;
  hipStreamCreate(&stream1);

  o_edge*       h_edges;
  o_edge*       v_edges;
  int*          h_idx;
  int*          v_idx;
  int*          cells_d;
  int*          mbrs_d;
  check_result* results;
  hipMallocAsync((void**)&h_edges, sizeof(o_edge) * hes.size(), stream1);
  hipMallocAsync((void**)&v_edges, sizeof(o_edge) * ves.size(), stream1);
  hipMemcpyAsync(h_edges, hes.data(), sizeof(o_edge) * hes.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(v_edges, ves.data(), sizeof(o_edge) * ves.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMallocAsync((void**)&h_idx, sizeof(int) * hidx.size(), stream1);
  hipMallocAsync((void**)&v_idx, sizeof(int) * vidx.size(), stream1);
  hipMemcpyAsync(h_idx, hidx.data(), sizeof(int) * hidx.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(v_idx, vidx.data(), sizeof(int) * vidx.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMallocAsync((void**)&mbrs_d, sizeof(int) * mbrs.size(), stream1);
  hipMemcpyAsync(mbrs_d, mbrs.data(), sizeof(int) * mbrs.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMallocAsync((void**)&cells_d, sizeof(int) * cells.size(), stream1);
  hipMemcpyAsync(cells_d, cells.data(), sizeof(int) * cells.size(),
                  hipMemcpyHostToDevice, stream1);
  hipMallocAsync((void**)&results, sizeof(check_result) * 150000, stream1);

  auto rows = layout_partition(db, std::vector{layer1}, threshold);
  hipStreamSynchronize(stream1);

  std::vector<evnt> events;
  std::vector<int>  eidx;
  events.reserve(cells.size() * 2);
  eidx.reserve(rows.size() + 1);
  eidx.emplace_back(0);
  evnt* events_d = nullptr;
  int*  eidx_d   = nullptr;
  hipMallocAsync((void**)&events_d, cells.size() * sizeof(evnt) * 2, stream1);
  hipMallocAsync((void**)&eidx_d, (rows.size() + 1) * sizeof(int), stream1);
  int bs = 128;
  for (int i = 0; i < rows.size(); ++i) {
    // if(i != 49) continue;
    int bs          = 128;
    int rsize       = rows[i].size();
    int total_check = rsize * (rsize + 1) / 2;

    int start_offset = events.size();
    for (int j = 0; j < rows[i].size(); ++j) {
      int* mbrj = &mbrs[rows[i][j] * 4];
      events.emplace_back(evnt{mbrj[0], mbrj[2], mbrj[3], rows[i][j] + 1});
      events.emplace_back(evnt{mbrj[1], mbrj[2], mbrj[3], -rows[i][j] - 1});
    }
    int end_offset = events.size();
    eidx.emplace_back(end_offset);
    hipStreamSynchronize(stream1);
    hipMemcpyAsync(events_d + start_offset, &events[start_offset],
                    sizeof(evnt) * (end_offset - start_offset),
                    hipMemcpyHostToDevice, stream1);

    thrust::async::sort(thrust::device, events_d + start_offset,
                        events_d + end_offset,
                        [] __device__(const auto& e1, const auto& e2) {
                          return e1.x == e2.x ? e1.id > e2.id : e1.x < e2.x;
                        });
  }
  hipMemcpy(eidx_d, eidx.data(), sizeof(int) * eidx.size(),
             hipMemcpyHostToDevice);
  bs = 32;
  run_row<<<((rows.size() + 1) / 2 + bs - 1), bs>>>(
      h_edges, v_edges, h_idx, v_idx, mbrs_d, results, events_d, eidx_d,
      int(eidx.size()), threshold, 0);
  run_row<<<((rows.size() + 1) / 2 + bs - 1), bs>>>(
      h_edges, v_edges, h_idx, v_idx, mbrs_d, results, events_d, eidx_d,
      int(eidx.size()), threshold, 1);
  hipDeviceSynchronize();
  result_transform(vios, results, sizeof(results) / sizeof(check_result));
}
}  // namespace odrc